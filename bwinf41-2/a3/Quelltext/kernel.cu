
#include <hip/hip_runtime.h>
#include <iostream>

using u8 = unsigned char;
using u64 = unsigned long long;
struct Bestes
{
    u8 laenge;
    u8 letzter_flip;
};
struct BestesMitEnumeration
{
    u64 enumeration;
    Bestes bestes;
};

inline __device__ u64 enumerate_permutation(u8 *permutation, u8 *indeces, u8 len)
{
    for (u8 i = 0; i < len; i++)
    {
        u8 index = (u8)-1;
        for (u8 j = 0; j < len - i; j++)
        {
            if (index != (u8)-1)
                permutation[j - 1] = permutation[j];
            else if (permutation[j] - 1 == i)
                index = j;
        }
        if (index == (u8)-1)
            return (u64)-1;
        indeces[i] = index;
    }
    u64 result = 0;
    u64 fact = 1;
    for (u8 i = 0; i < len; i++)
    {
        if (i > 1)
            fact *= i;
        result += indeces[i] * fact;
    }
    return result;
}

inline __device__ void permutation_by_enumeration(u64 enumeration, u8 *result, u8 *indeces, u8 len, u64 fact)
{
    for (u8 i = 0; i < len; i++)
    {
        fact /= len - i;
        indeces[i] = enumeration / fact;
        enumeration %= fact;
    }
    for (u8 i = len - 1; i < len; i--)
    {
        u8 index = indeces[i];
        memcpy(result + index + 1, result + index, sizeof(u8));
        result[index] = i + 1;
    }
}

inline __device__ void wenden_und_essen(u8 *stapel, u8 *neuer_stapel, u8 len, u8 index)
{
    u8 gegessen = stapel[index];
    for (u8 i = 0; i < index; i++)
    {
        u8 pfannkuchen = stapel[i];
        if (pfannkuchen > gegessen)
            pfannkuchen--;
        neuer_stapel[i] = pfannkuchen;
    }
    for (u8 i = 0; i < len - index - 1; i++)
    {
        u8 pfannkuchen = stapel[len - 1 - i];
        if (pfannkuchen > gegessen)
            pfannkuchen--;
        neuer_stapel[index + i] = pfannkuchen;
    }
}

extern "C" __global__ void run_permutations(Bestes *prior, Bestes *current, BestesMitEnumeration *bestes_gefundene, u8 size, u64 fact)
{
    printf("Fact: %llu\n", fact);
    u64 index = blockIdx.x * blockDim.x + threadIdx.x;
    u64 max_elements = fact / (blockDim.x * gridDim.x) + 1;
    if (index + max_elements >= fact)
        max_elements = fact - index; // check if > or >=

    u64 enumeration;
    u8 indeces[16];
    u8 result[16];
    u8 neuer_stapel_tmp[16];
    Bestes momentan_bestes;
    bestes_gefundene[index].bestes.laenge = (u8)-1;

    for (u64 i = 0; i < max_elements; i++)
    {
        enumeration = index + i;
        permutation_by_enumeration(enumeration, result, indeces, size, fact);
        momentan_bestes.laenge = (u8)-1;

        for (u8 flip = 0; flip < size; flip++)
        {
            wenden_und_essen(result, neuer_stapel_tmp, size, flip);
            u64 neue_permutation = enumerate_permutation(neuer_stapel_tmp, indeces, size - 1);

            Bestes potenziel_bestes = prior[neue_permutation];
            if (momentan_bestes.laenge > potenziel_bestes.laenge)
                momentan_bestes = potenziel_bestes;
        }
        momentan_bestes.laenge++;
        if (momentan_bestes.laenge != 0)
            current[enumeration] = momentan_bestes;
        if (momentan_bestes.laenge < bestes_gefundene[index].bestes.laenge)
        {
            bestes_gefundene[index].bestes = momentan_bestes;
            bestes_gefundene[index].enumeration = enumeration;
        }
    }
}