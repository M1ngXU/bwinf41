
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdarg.h>

using u8 = unsigned char;
using u64 = unsigned long long;
struct Bestes
{
    u8 laenge;
    u8 letzter_flip;
};
struct BestesMitEnumeration
{
    u64 enumeration;
    Bestes bestes;
};

template <typename... Args>
inline __device__ void print(const char *f, Args... args)
{
    // printf(f, args...);
}

inline __device__ u64 enumerate_permutation(u8 *permutation, u8 *indeces, u8 len)
{
    print("Permutation: ");
    for (u8 i = 0; i < len; i++)
        print("%u|", permutation[i]);
    print("\n");
    for (u8 i = 0; i < len; i++)
    {
        u8 index = (u8)-1;
        for (u8 j = 0; j < len - i; j++)
        {
            if (index != (u8)-1)
                permutation[j - 1] = permutation[j];
            else if (permutation[j] - 1 == i)
                index = j;
        }
        assert(index != (u8)-1);
        indeces[i] = index;
    }
    print("Indeces: ");
    for (u8 i = 0; i < len; i++)
        print("%u|", indeces[i]);
    print("\n");
    u64 result = 0;
    u64 fact = 1;
    for (u8 i = 0; i < len; i++)
    {
        if (i > 1)
            fact *= i;
        result += indeces[len - 1 - i] * fact;
    }
    print("Enumeration: %u\n", result);
    return result;
}

inline __device__ void permutation_by_enumeration(u64 enumeration, u8 *result, u8 *indeces, u8 len, u64 fact)
{
    for (u8 i = 0; i < len; i++)
    {
        fact /= len - i;
        indeces[i] = enumeration / fact;
        enumeration %= fact;
    }
    for (u8 i = len - 1; i < len; i--)
    {
        u8 index = indeces[i];
        for (u8 j = len - i - 1; j > index; j--)
            result[j] = result[j - 1];
        result[index] = i + 1;
    }
}

inline __device__ void wenden_und_essen(u8 *stapel, u8 *neuer_stapel, u8 len, u8 index)
{
    print("Vorher: ");
    for (u8 i = 0; i < len; i++)
        print("%u|", stapel[i]);
    print("\n");
    u8 gegessen = stapel[index];
    for (u8 i = 0; i < index; i++)
    {
        u8 pfannkuchen = stapel[i];
        if (pfannkuchen > gegessen)
            pfannkuchen--;
        neuer_stapel[i] = pfannkuchen;
    }
    for (u8 i = 0; i < len - index - 1; i++)
    {
        u8 pfannkuchen = stapel[len - 1 - i];
        if (pfannkuchen > gegessen)
            pfannkuchen--;
        neuer_stapel[index + i] = pfannkuchen;
    }
    print("Nachher: ");
    for (u8 i = 0; i < len - 1; i++)
        print("%u|", neuer_stapel[i]);
    print("\n");
}

// https://en.wikipedia.org/wiki/Permutation#Generation_in_lexicographic_order
inline __device__ void permutate(u8 *stapel, u8 len, u64 enumeration)
{
    u8 k = len - 2;
    while (k <= len - 2 && stapel[k] > stapel[k + 1])
        k--;
    if (k == (u8)-1)
    {
        print("%llu|%u|", enumeration, len);
        for (u8 i = 0; i < len; i++)
            print("%u|", stapel[i]);
        print("\n");
        assert(false);
    }

    u8 i = len - 1;
    while (stapel[k] > stapel[i])
        i--;

    u8 tmp = stapel[k];
    stapel[k] = stapel[i];
    stapel[i] = tmp;

    u8 swap_count = (len - k) / 2;
    for (i = 0; i < swap_count; i++)
    {
        u8 tmp = stapel[k + 1 + i];
        stapel[k + 1 + i] = stapel[len - 1 - i];
        stapel[len - 1 - i] = tmp;
    }
}

extern "C" __global__ void run_permutations(Bestes *prior, Bestes *current, BestesMitEnumeration *bestes_gefundene, u8 size, u64 fact)
{
    u64 max_elements = fact / (blockDim.x * gridDim.x) + 1;
    u64 index = blockIdx.x * blockDim.x + threadIdx.x;
    u64 enumeration = index * max_elements;
    if (enumeration + max_elements >= fact)
        max_elements = fact - enumeration; // check if > or >=

    printf("%llu|%llu|%llu|%llu\n", max_elements, index, enumeration, fact);
    u8 indeces[16];
    u8 result[16];
    permutation_by_enumeration(enumeration, result, indeces, size, fact);

    u8 neuer_stapel_tmp[16];
    Bestes momentan_bestes;
    bestes_gefundene[index].bestes.laenge = 0;

    for (u64 i = 0; i < fact / size; i++)
    {
        print("%llu: %u (%u)\n", i, prior[i].laenge, prior[i].letzter_flip);
    }

    for (u64 i = 0; i < max_elements; i++)
    {
        if (i > 0)
        {
            // permutation_by_enumeration(++enumeration, result, indeces, size, fact);
            permutate(result, size, enumeration++);
        }

        momentan_bestes.letzter_flip = (u8)-1;
        bool sortiert = true;
        u8 letztes = (u8)-1;
        for (u8 j = 0; sortiert && j < size; j++)
        {
            if (result[j] > letztes)
                sortiert = false;
            else
                letztes = result[j];
        }

        if (sortiert)
            momentan_bestes.laenge = 0;
        else
        {
            momentan_bestes.laenge = (u8)-1;
            for (u8 flip = 0; !sortiert && flip < size; flip++)
            {
                wenden_und_essen(result, neuer_stapel_tmp, size, flip);

                sortiert = true;
                letztes = (u8)-1;
                for (u8 j = 0; sortiert && j < size - 1; j++)
                {
                    if (neuer_stapel_tmp[j] > letztes)
                        sortiert = false;
                    else
                        letztes = neuer_stapel_tmp[j];
                }
                u8 potenziel_beste_laenge = 1;
                if (!sortiert)
                {
                    u64 neue_enumeration = enumerate_permutation(neuer_stapel_tmp, indeces, size - 1);
                    u8 vorherige_laenge = prior[neue_enumeration].laenge;
                    potenziel_beste_laenge += vorherige_laenge;
                    print("%llu|%u: %u vs %u (%llu:%u)\n", enumeration, flip, potenziel_beste_laenge, momentan_bestes.laenge, neue_enumeration, vorherige_laenge);
                }
                if (potenziel_beste_laenge < momentan_bestes.laenge)
                {
                    momentan_bestes.laenge = potenziel_beste_laenge;
                    momentan_bestes.letzter_flip = flip;
                }
            }
        }
        if (momentan_bestes.laenge != (u8)-1)
        {
            current[enumeration] = momentan_bestes;
            if (momentan_bestes.laenge > bestes_gefundene[index].bestes.laenge)
            {
                bestes_gefundene[index].bestes = momentan_bestes;
                bestes_gefundene[index].enumeration = enumeration;
            }
        }
    }
}